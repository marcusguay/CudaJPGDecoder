#include "hip/hip_runtime.h"
#include "./include/GPUDecoder.hpp"

#define CHECK_CUDA_ERROR(err)                                            \
    if (err != hipSuccess)                                              \
    {                                                                    \
        fprintf(stderr, "CUDA Error at %s:%d: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err));                                \
        return;                                                          \
    }

// Device variables
float *(d_blocks);
QuantizationTable(*d_quantizationTables);
int *(d_zigzagTable);
ImageSpecification *d_imageSpecification;

// Host variables
ImageSpecification imageSpecification;

__global__ void zigzagScanCUDA(float *d_matrix, int *d_zigzagTable)
{
    /* Converts a matrix into its zigzag ordered matrix */

    __shared__ float temp[64];
    int idx = threadIdx.x + (threadIdx.y * 8) + (blockIdx.x * 64);
    int zigZagIndex = threadIdx.x + threadIdx.y * 8;
    temp[zigZagIndex] = d_matrix[idx];

    __syncthreads();
    d_matrix[idx] = temp[d_zigzagTable[zigZagIndex]];
}

__global__ void check(float *d_matrix)
{
    int idx = threadIdx.x + (threadIdx.y * 8) + (blockIdx.x * 64);
    printf("at %d is %d \n", idx, d_matrix[idx]);
}

__global__ void applyQuantizationTables(float *d_matrix, QuantizationTable *d_quantizationTables, int *quantizationIndices)
{
    /* Applies element wise multiplication with correct quantization table for component  */

    int qtableIndex = quantizationIndices[((blockIdx.x % 3) + 1)];
    int insideQTTableIdx = threadIdx.x + (threadIdx.y * 8);
    int idx = threadIdx.x + (threadIdx.y * 8) + (blockIdx.x * 64);
    QuantizationTable table = d_quantizationTables[qtableIndex];

    if (table.precision == 0)
    {

        d_matrix[idx] = d_matrix[idx] * table.table8[insideQTTableIdx];
    }
    else
    {
        d_matrix[idx] = d_matrix[idx] * table.table16[insideQTTableIdx];
    }

    __syncthreads();
}

__global__ void inverseDCT(float *d_matrix)
{
    /* Standard formula for 2D Inverse Discrete Cosine Transform
       there are probably more optimized versions of this that I should probably look into.... */

    __shared__ float temp[64];
    int offset = (blockIdx.x * 64);
    int idx = threadIdx.x + (threadIdx.y * 8) + offset;
    int localIndex = threadIdx.x + (threadIdx.y * 8);
    temp[localIndex] = 0.0f;

    for (int i = 0; i < 8; i++)
    {
        for (int j = 0; j < 8; j++)
        {

            float ci = (i == 0) ? 1.0 / sqrtf(2.0) : 1.0;
            float cj = (j == 0) ? 1.0 / sqrtf(2.0) : 1.0;
            int inputIndex = j * 8 + i + offset;
            temp[localIndex] += ci * cj * d_matrix[inputIndex] *
                                cosf((2.0 * threadIdx.x + 1.0) * i * M_PI / 16.0) *
                                cosf((2.0 * threadIdx.y + 1.0) * j * M_PI / 16.0);
        }
    }

    __syncthreads();

    /* JPG specs say to divide by four after the IDCT */
    d_matrix[idx] = roundf(temp[localIndex] / 4.0f);
}

__global__ void YCbCrToRGB(float *d_matrix, RGB *pixels, int imageWidth, int imageHeight)
{

    /*  Index translation here gets a bit complicated since the indices we
        are working with in the float array dont correspond to the indices the
        BMP file is expecting, ie: index 8 corresponds to (0, 1) in our 8 x 8 blocks
        yet index 8 in the BMP image it would be (9, 0)
    */

    int blockX = blockIdx.x % ((imageWidth + 7) / 8); // Find which CUDA block we are in
    int blockY = blockIdx.x / ((imageWidth + 7) / 8);

    int offsetX = blockX * 8 + threadIdx.x; // each X block index contributes 8 to the offset + our local x offset
                                            /*
                                            This one is the one that is hard to see until, you draw out the mappings
                                            For each thread Y index it adds the image width to the offset, each block index is equivalent
                                            to increasing the thread Y index by 8
                                            */


    /*
    Checks if padding pixels are contributing to final image
    */
    if (offsetX >= imageWidth || (blockY * 8 + threadIdx.y) >= imageHeight) {
        return;
    }


    int offsetY = (blockY * 8 + threadIdx.y) * imageWidth;
    int pixelIndex = offsetX + offsetY;

    /* We are expecting the MCU pattern to be (Y, Cb, Cr) consecutively */
    int yIndex = threadIdx.x + (threadIdx.y * 8) + blockIdx.x * 192;
    int cbIndex = yIndex + 64;
    int crIndex = yIndex + 128;

    /* Level only shift for luminance */
    float Y = d_matrix[yIndex] + 128.0;
    float Cb = d_matrix[cbIndex];
    float Cr = d_matrix[crIndex];

    /* (Y, Cb, Cr) -> (R,G,B) conversion formula according to the JPG specification */
    float R = Y + 1.402f * Cr;
    float G = Y - 0.344136f * Cb - 0.714136f * Cr;
    float B = Y + 1.772f * Cb;

    /* Have to make sure the values are in the [0,255] range */
    R = static_cast<uint8_t>(roundf(fminf(fmaxf(R, 0.0f), 255.0f)));
    G = static_cast<uint8_t>(roundf(fminf(fmaxf(G, 0.0f), 255.0f)));
    B = static_cast<uint8_t>(roundf(fminf(fmaxf(B, 0.0f), 255.0f)));

    /* Save the pixel to the right index of the array*/
    
    if(blockX == 0){
       printf("%d %d PixelIDX %d \n", blockX, blockY, pixelIndex);
    }

    pixels[pixelIndex].r = R;
    pixels[pixelIndex].g = G;
    pixels[pixelIndex].b = B;
}

void decodeImageCuda(std::vector<float> &flatBlocks, int quantizationIndex[4])
{

    /* Only supports (Y, Cb, Cr) images for now... */
    int numberOfComponents = 3;

    int numBlocks = std::ceil(flatBlocks.size() / 64);
    int size = sizeof(float) * flatBlocks.size();

    /* Each MCU is 8 x 8 block */
    dim3 nThreads(8, 8);

    hipError_t err;

    int *quantizationIndices;

    std::cout << "flatBlocks size: " << flatBlocks.size() << ", total size in bytes: " << size << "\n";
    std::cout << "Quantization indexes " << quantizationIndex[1] << " " << quantizationIndex[2] << " " << quantizationIndex[3] << "\n";

    CHECK_CUDA_ERROR(hipMalloc(&d_blocks, size));
    CHECK_CUDA_ERROR(hipMemcpy(d_blocks, flatBlocks.data(), size, hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipMalloc(&quantizationIndices, sizeof(int) * 4));
    CHECK_CUDA_ERROR(hipMemcpy(quantizationIndices, quantizationIndex, sizeof(int) * 4, hipMemcpyHostToDevice));

    std::cout << "allocated " << numBlocks << " of size 64 on GPU " << "\n";

    applyQuantizationTables<<<numBlocks, nThreads>>>(d_blocks, d_quantizationTables, quantizationIndices);
    hipDeviceSynchronize();

    err = hipGetLastError();
    CHECK_CUDA_ERROR(err);

    zigzagScanCUDA<<<numBlocks, nThreads>>>(d_blocks, d_zigzagTable);
    hipDeviceSynchronize();

    err = hipGetLastError();
    CHECK_CUDA_ERROR(err);

    inverseDCT<<<numBlocks, nThreads>>>(d_blocks);
    hipDeviceSynchronize();

    err = hipGetLastError(); // is there a better to check this, then to do it for each CUDA call?
    CHECK_CUDA_ERROR(err);

    RGB *d_pixels;
    int numPixels = imageSpecification.width * imageSpecification.height;
    std::cout << "image has num pixels " << numPixels << "\n";
    CHECK_CUDA_ERROR(hipMalloc(&d_pixels, sizeof(RGB) * numPixels));
    std::cout << "num 192 blocks " << (numBlocks / numberOfComponents) << "\n";

    /* Grid and block calculation here are a bit different
      since we are combining three components into one pixel
      we should launch a third of the blocks with size 192, (3 * 64)
      instead of the usual 64 size */

    int numMCUBlocks = numBlocks / numberOfComponents;
    int numBlocksWidth = (imageSpecification.width + 7) / 8;

    YCbCrToRGB<<<numMCUBlocks, nThreads>>>(d_blocks, d_pixels, imageSpecification.width, imageSpecification.height);
    hipDeviceSynchronize();

    std::vector<RGB> rgb(numPixels);
    CHECK_CUDA_ERROR(hipMemcpy(rgb.data(), d_pixels, sizeof(RGB) * numPixels, hipMemcpyDeviceToHost));

    writeBMP("/usr/src/app/testImagesOutput/output1.bmp", rgb, imageSpecification.width, imageSpecification.height);
    std::cout << "decoded image to BMP at /usr/src/app/testImagesOutput !" << "\n";

    /* Now we have to clear everything from GPU memory
       for now the program is supposed to be ran once per image
       decoded, so clear zigzagtable too */

    hipFree(d_blocks);
    hipFree(d_imageSpecification);
    hipFree(d_pixels);
    hipFree(d_quantizationTables);
    hipFree(d_zigzagTable);
}

void setupCuda(QuantizationTable (&quantizationTables)[4], ImageSpecification imageSpecificationHost)
{

    hipError_t err;
    imageSpecification = imageSpecificationHost;

    err = hipDeviceReset();
    CHECK_CUDA_ERROR(err);

    err = hipSetDevice(0);
    CHECK_CUDA_ERROR(err);

    /* print device properties
       (to see if GPU is available to use) */

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, 0);
    CHECK_CUDA_ERROR(err);
    printf("Using GPU: %s\n", prop.name);

    /* allocate all the stuff we need on GPU */
    int quantizationTableSize = sizeof(QuantizationTable) * 4;
    CHECK_CUDA_ERROR(hipMalloc(&d_quantizationTables, quantizationTableSize));
    CHECK_CUDA_ERROR(hipMemcpy(d_quantizationTables, quantizationTables, quantizationTableSize, hipMemcpyHostToDevice));

    int zigZagTableSize = sizeof(int) * 64;
    CHECK_CUDA_ERROR(hipMalloc(&d_zigzagTable, zigZagTableSize));
    CHECK_CUDA_ERROR(hipMemcpy(d_zigzagTable, zigzagTable, zigZagTableSize, hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipMalloc(&d_imageSpecification, sizeof(ImageSpecification)));
    CHECK_CUDA_ERROR(hipMemcpy(d_imageSpecification, &imageSpecificationHost, sizeof(ImageSpecification), hipMemcpyHostToDevice));
}
